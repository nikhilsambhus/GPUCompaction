
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <assert.h>
#include <pthread.h>
#include <stdint.h>
#define MAX 1024*1024*1024
#define N_STREAMS 4
#define N_THREADS 8
#define KSIZE 16
//#define VSIZE (512-16)
#define VSIZE (32-16)
//#define VSIZE (128-16)

typedef struct {
	char key[KSIZE];
	char value[VSIZE];
} _kv;

typedef struct {
	_kv **inp1;
	_kv **inp2;
	_kv **out;
	int psize;
	int noparts;
} _compdata;

struct args {
	_compdata *data;
	int t_no;
};

struct partStart {
	int ai, bi;
};
//str functions for cuda
__device__ void c_strcpy(char *dest, char *src) {
	int i = 0;
	while(src[i] != '\0') {
		dest[i] = src[i];
		i++;
	}
	dest[i] = '\0';
}

__device__ void c_memcpy(void *dest, void *src, int size) {
	//memcpy(dest, src, size);
	//return;
	int i;
	uint64_t *dp = (uint64_t *)dest, *sp = (uint64_t *)src;
	for(i = 0; i < size; i+=8) {
		*dp = *sp;
	}
}	
__device__ int c_atoi(char *str) {
	int i = 0, ret = 0;
	while(str[i] != '\0') {
		ret = ret * 10 + str[i] - '0';
		i++;
	}
	return ret;
}

double getTime(void) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (double)tv.tv_usec/1e6 + tv.tv_sec;
}

void checkStatus(hipError_t st, const char *method) {
	if(st != hipSuccess) {
		printf("Error: Error while doing %s\n", method);
		exit(-1);
	}
}

void cpuMerge(_kv *a, _kv *b, _kv *res, int size) {
	int i = 0, j = 0, z = 0;
	while (z < size * 2) {
		if((i < size) && (j < size)) {
			if(atoi(a[i].key) <= atoi(b[j].key)) {
				memcpy(&res[z], &a[i], KSIZE+VSIZE);
				i++;
			}
			else {
				memcpy(&res[z], &b[j], KSIZE+VSIZE);
				j++;
			}
		}
		else if(i < size) {
			memcpy(&res[z], &a[i], KSIZE+VSIZE);
			i++;
		}
		else if (j < size) {
			memcpy(&res[z], &b[j], KSIZE+VSIZE);
			j++;
		}
		z++;
	}
}

void* tCpuMerge(void *args) {
	int i;
	struct args *part = (struct args *)args;
	int chunksize = part->data->noparts/N_THREADS;
	for(i = part->t_no * chunksize; i < (part->t_no + 1) * chunksize; i++) {
		cpuMerge(part->data->inp1[i], part->data->inp2[i], part->data->out[i], part->data->psize);
	}
	return NULL;
}
__device__ int binSearch(char *elem, _kv *Sa, int n, int *is_mid) {

	int low = 0, high = n - 1, mid;
	*is_mid = 0;
	int elem_i, mid_i;
	elem_i = c_atoi(elem);
	if(elem_i > c_atoi(Sa[high].key)) {
		return high + 1;
	}
	else if(elem_i < c_atoi(Sa[low].key)) {
		return low;
	}
	while(low <= high) {
		mid = (low + high)/2;
		elem_i = c_atoi(elem);  
		mid_i = c_atoi(Sa[mid].key);
		if(elem_i < mid_i) {
			high = mid - 1;
		}
		else if(elem_i > mid_i) {
			low = mid + 1;
		}
		else {
			*is_mid = 1;
			break;
		}
	}
	
	return mid;
}

__global__ void d_binS_merge(_kv *A, _kv *B, _kv *C, int n) {
	int pos = blockIdx.x * blockDim.x + threadIdx.x;
	int is_mid = 0;
	int ret;
	//find final location for A[pos] and B[pos]
	if(pos < n) {
		ret = 0;
		ret = binSearch(A[pos].key, B, n, &is_mid);
		c_strcpy(C[pos + ret + is_mid].key, A[pos].key);
		memcpy(C[pos + ret + is_mid].value, A[pos].value, VSIZE);

		ret = binSearch(B[pos].key, A, n, &is_mid);
		c_strcpy(C[pos + ret].key, B[pos].key);
		memcpy(C[pos + ret].value, A[pos].value, VSIZE);
	}

}


__device__ struct partStart diagInter(_kv *A, _kv *B, int n, int pindex, int parts) {
	int diag = ((n/parts) << 1) * pindex;
	int begin = ((diag - n ) > 0 ? diag - n: 0);
	int end = (diag > n ? n : diag);
	int aKey, bKey;
	struct partStart ABstart;
	while(begin < end) {
		int mid = (begin + end) >> 1;
		aKey = c_atoi(A[mid].key);
		bKey = c_atoi(B[diag - 1 - mid].key);
		if(aKey < bKey) {
			begin = mid + 1;
		}
		else {
			end = mid;
		}
	}

	ABstart.ai = begin;
	ABstart.bi = diag - begin;
	//printf("pindex %d ai %d bi %d\n", pindex, ABstart.ai, ABstart.bi);
	return ABstart;

}

__device__ void merge(_kv *A, int ai, _kv *B, int bi, _kv *C, int ci, int n, int len) {
	int i = ai, j = bi, z = ci;
	while (z < ci + len) {
		if((i < n) && (j < n)) {
			if(c_atoi(A[i].key) <= c_atoi(B[j].key)) {
				c_memcpy(&C[z], &A[i], KSIZE + VSIZE);
				i++;
			}
			else {
				c_memcpy(&C[z], &B[j], KSIZE + VSIZE);
				j++;
			}
		}
		else if(i < n) {
			c_memcpy(&C[z], &A[i], KSIZE + VSIZE);
			i++;
		}
		else if (j < n) {
			c_memcpy(&C[z], &B[j], KSIZE + VSIZE);
			j++;
		}
		z++;
	}
}
__global__ void d_pathMerge(_kv *A, _kv *B, _kv *C, int n, int parts) {
	//return;
	int lenPart = ((n << 1)/parts);
	//lenPart = 0;
	struct partStart ABstart;
	int Cstart;
	int pindex = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(pindex < parts) {
		ABstart = diagInter(A, B, n, pindex, parts);
		Cstart = (n << 1)/parts * pindex;
		merge(A, ABstart.ai, B, ABstart.bi, C, Cstart, n, lenPart);
	}
}
void pathMerge(_kv *A, _kv *B, _kv *C, int n, int parts, hipStream_t stream) {
	dim3 block(32);
	dim3 grid((parts + block.x - 1)/block.x);
	d_pathMerge<<<grid, block, 0, stream>>> (A, B, C, n, parts);
}
int check_sorted(_compdata *data) {
	int i, j;
	for(i = 0; i < data->noparts; i++) {
		for(j = 0; j < (2 * data->psize) - 1; j++) {
			//printf("%d %s\n", j, data->out[i][j].key);
			if(atoi(data->out[i][j].key) > atoi(data->out[i][j+1].key)) {
				printf("Value incorrect at pos %d %s\n", j + 1, data->out[i][j+1].key);
				return 0;
			}
		}
		//printf("%d %s\n", j, data->out[i][j].key);
	}
	return 1;
}

int cmpF(const void *a, const void *b) {
	int v1 = *(int *)a;
	int v2 = *(int *)b;
	if(v1 < v2) return -1;
	else if(v1 > v2) return 1;
	else return 0;
}	

void initCompData(_compdata *data) {
	int i, j;
	int bytes = data->psize * sizeof(_kv);
	int *randSet1, *randSet2;
	randSet1 = (int *)malloc(data->psize * sizeof(int));
	randSet2 = (int *)malloc(data->psize * sizeof(int));
	srand(100);
	for (i = 0; i < data->psize; i++) {
		randSet1[i] = rand();
		randSet2[i] = rand();
	}
	qsort(randSet1, data->psize, sizeof(int), cmpF);
	qsort(randSet2, data->psize, sizeof(int), cmpF);
	hipError_t status;
	data->inp1 = (_kv **) malloc(sizeof(_kv *) * data->noparts);
	data->inp2 = (_kv **) malloc(sizeof(_kv *) * data->noparts);
	data->out = (_kv **) malloc(sizeof(_kv *) * data->noparts);
	for(i = 0; i < data->noparts; i++) {
		status = hipHostAlloc((void **)&data->inp1[i], bytes, hipHostMallocDefault);
		checkStatus(status, "hipHostAlloc");
		status = hipHostAlloc((void **)&data->inp2[i], bytes, hipHostMallocDefault);
		checkStatus(status, "hipHostAlloc");
		status = hipHostAlloc((void **)&data->out[i], 2 * bytes, hipHostMallocDefault);
		checkStatus(status, "hipHostAlloc");
		for(j = 0; j < data->psize; j++) {
			sprintf(data->inp1[i][j].key, "%015d", randSet1[j]);
			memset(data->inp1[i][j].value, 'x', VSIZE);
			sprintf(data->inp2[i][j].key, "%015d", randSet2[j]);
			memset(data->inp2[i][j].value, 'x', VSIZE);
			//printf("%s %s\n", data->inp1[i][j].key, data->inp2[i][j].key);
		}
	}

	free(randSet1);
	free(randSet2);
	printf("Initilization done\n");
	return;
}

void deviceMalloc(_kv **d_inp1, _kv **d_inp2, _kv **d_out, int psize) {
	int i;
	hipError_t status;
	for(i = 0; i < N_STREAMS; i++) {
		status = hipMalloc((void **)&d_inp1[i], psize * sizeof(_kv));
		checkStatus(status, "hipMalloc");
		status = hipMalloc((void **)&d_inp2[i], psize * sizeof(_kv));
		checkStatus(status, "hipMalloc");
		status = hipMalloc((void **)&d_out[i], 2 * psize * sizeof(_kv));
		checkStatus(status, "hipMalloc");
	}

}
int main(int argc, char *argv[])
{
	hipError_t status;
	_compdata data;	
	struct args part[N_THREADS];
	pthread_t tids[N_THREADS];
	int i, j;
	int bytes;
	_kv *d_inp1[N_STREAMS], *d_inp2[N_STREAMS], *d_out[N_STREAMS];
	hipStream_t streams[N_STREAMS];
	double ts_a, ts_b, ts_c, ts_d;

	if(argc != 3) {
		printf("Required: partition size, number of partitions\n");
		return -1;
	}
	data.psize = atoi(argv[1]);
	data.noparts = atoi(argv[2]);
	bytes = data.psize * sizeof(_kv);
	printf("Total size (2 inputs + 1 output) %.2f MB\n", (bytes * 4.0 * data.noparts)/1024/1024);
	printf("Size of one partition (2 inputs + 1 output) %.2f MB\n", (bytes * 4.0)/1024/1024);
	//initilization
	initCompData(&data);
	deviceMalloc(d_inp1, d_inp2, d_out, data.psize);
	dim3 block(1024);
	dim3 grid((data.psize + block.x - 1)/block.x);

	//create streams
	for(i = 0; i < N_STREAMS; i++) {
		hipStreamCreate(&streams[i]);
	}

	assert(data.noparts % N_STREAMS == 0);
	ts_a = getTime();
	//launch transfer and kernels in streams
	for(i = 0; i < data.noparts; i += N_STREAMS) {
		for(j = 0; j < N_STREAMS; j++) {
			hipMemcpyAsync(d_inp1[j], data.inp1[i+j], bytes, hipMemcpyHostToDevice, streams[j]);
			hipMemcpyAsync(d_inp2[j], data.inp2[i+j], bytes, hipMemcpyHostToDevice, streams[j]);
			//launch kernel
			//d_binS_merge <<<grid, block, 0, streams[j]>>> (d_inp1[j], d_inp2[j], d_out[j], data.psize);
			pathMerge(d_inp1[j], d_inp2[j], d_out[j], data.psize, (data.psize << 1), streams[j]); 
			hipMemcpyAsync(data.out[i+j], d_out[j], 2 * bytes, hipMemcpyDeviceToHost, streams[j]);
		}

	}
	status = hipDeviceSynchronize();
	ts_b = getTime();
	checkStatus(status, "cudaDeviceSync");
	if(check_sorted(&data) == 0) {
		printf("Result GPU incorrect\n");
	}

	ts_c = getTime();
	for(i = 0; i < N_THREADS; i++) {
		part[i].data = &data;
		part[i].t_no = i;
		pthread_create(&tids[i], NULL, &tCpuMerge, (void *)&part[i]);
	}
	for(i = 0; i < N_THREADS; i++) {
		pthread_join(tids[i], NULL);
	}
	ts_d = getTime();
	if(check_sorted(&data) == 0) {
		printf("Result CPU incorrect\n");
	}
	printf("Execution time Streaming %d GPU %.6f %d threads CPU %.6f\n",N_STREAMS, ts_b - ts_a, N_THREADS, ts_d - ts_c);

	hipDeviceReset();

	return 0;
}
